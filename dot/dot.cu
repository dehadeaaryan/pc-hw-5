
#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include <chrono>
#include <random>
#include <cstdlib>

#if !defined(__CUDA_ARCH__) || __CUDA_ARCH__ >= 600
#else
__device__ double atomicAdd(double* address, double val) {
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}
#endif

// CUDA Kernel to compute dot product
__global__ void dotProduct(double *a, double *b, double *result, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    __shared__ double temp[256];

    temp[threadIdx.x] = 0;

    for (int i = index; i < n; i += stride) {
        temp[threadIdx.x] += a[i] * b[i];
    }

    __syncthreads();

    // Reduction in shared memory
    for (int i = blockDim.x / 2; i > 0; i >>= 1) {
        if (threadIdx.x < i) {
            temp[threadIdx.x] += temp[threadIdx.x + i];
        }
        __syncthreads();
    }

    // Write the final sum to global memory
    if (threadIdx.x == 0) {
        atomicAdd(result, temp[0]);
    }
}

// CPU function to compute dot product
double dotProductCPU(std::vector<double>& a, std::vector<double>& b) {
    double result = 0.0;
    for (int i = 0; i < a.size(); ++i) {
        result += a[i] * b[i];
    }
    return result;
}

int main(int argc, char *argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <numBlocks> <threadsPerBlock>" << std::endl;
        return 1;
    }

    int numBlocks = atoi(argv[1]);
    int threadsPerBlock = atoi(argv[2]);

    int n = pow(2, 18);
    std::vector<double> a(n); // Initialize vector 'a' with random values
    std::vector<double> b(n); // Initialize vector 'b' with random values

    // Fill vectors with random values
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dis(1.0, 10.0); // Generate random values between 1 and 10
    for (int i = 0; i < n; ++i) {
        a[i] = dis(gen);
        b[i] = dis(gen);
    }

    double *d_a, *d_b, *d_result;
    double result = 0.0;

    hipError_t cudaStatus;

    cudaStatus = hipMalloc(&d_a, n * sizeof(double));
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed for d_a!" << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_b, n * sizeof(double));
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed for d_b!" << std::endl;
        return 1;
    }

    cudaStatus = hipMalloc(&d_result, sizeof(double));
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed for d_result!" << std::endl;
        return 1;
    }

    // Copy input data to device memory
    hipMemcpy(d_a, a.data(), n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), n * sizeof(double), hipMemcpyHostToDevice);

    // Benchmark CUDA implementation
    auto start = std::chrono::high_resolution_clock::now();
    dotProduct<<<numBlocks, threadsPerBlock>>>(d_a, d_b, d_result, n);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching dotProduct kernel!" << std::endl;
        return 1;
    }
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "Num Blocks: " << numBlocks << ", Threads Per Block: " << threadsPerBlock << ", CUDA time: " << duration.count() << " seconds" << std::endl;

    // Copy result back to host and print
    hipMemcpy(&result, d_result, sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "CUDA dot product: " << result << std::endl;

    // Benchmark CPU implementation
    start = std::chrono::high_resolution_clock::now();
    result = dotProductCPU(a, b);
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "CPU time: " << duration.count() << " seconds" << std::endl;
    std::cout << "CPU dot product: " << result << std::endl;

    // Free device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);

    return 0;
}